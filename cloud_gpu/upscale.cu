#include "hip/hip_runtime.h"
/**
 * Bicubic Image Upscaling Tool
 *
 * This program performs 2x bicubic upscaling on images using CUDA for GPU acceleration
 * or falls back to CPU with OpenMP parallelization. Bicubic interpolation provides
 * high-quality upscaling by considering a 4x4 neighborhood of pixels around each
 * target location and applying cubic interpolation in both x and y directions.
 *
 * Usage: ./upscaler [input_file] [output_file] [scale]
 */

/**
 * Bicubic Image Upscaling Tool
 *
 * This program performs 2x bicubic upscaling on images using CUDA for GPU acceleration
 * or falls back to CPU with OpenMP parallelization. Bicubic interpolation provides
 * high-quality upscaling by considering a 4x4 neighborhood of pixels around each
 * target location and applying cubic interpolation in both x and y directions.
 *
 * Usage: ./upscaler [input_file] [output_file] [scale]
 */

#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <exception>
#include <cstring>
#include <cmath>
#include <vector>
#ifdef _OPENMP
#include <omp.h>
#endif

// Clamp function for readability
__host__ __device__ float clamp(float value, float min_val, float max_val) {
    return min(max(value, min_val), max_val);
}

// Cubic interpolation function for 1D (Catmull-Rom spline)
__host__ __device__ float cubicInterpolate(float p0, float p1, float p2, float p3, float t) {
    return p1 + 0.5f * t * (2.0f * p0 - 5.0f * p1 + 4.0f * p2 - p3 + t * (3.0f * (p1 - p2) + p3 - p0));
}

// Perform bicubic interpolation on a pre-fetched 4x4 neighborhood
__host__ __device__ float perform_interpolation(const float vals[4][4], float tx, float ty) {
    float col[4];
    for (int m = 0; m < 4; m++) {
        col[m] = cubicInterpolate(vals[m][0], vals[m][1], vals[m][2], vals[m][3], tx);
    }
    float value = cubicInterpolate(col[0], col[1], col[2], col[3], ty);
    return fminf(fmaxf(value, 0.0f), 255.0f);
}
    float value = cubicInterpolate(col[0], col[1], col[2], col[3], ty);
    return clamp(value, 0.0f, 255.0f);
}

// Fetch the 4x4 neighborhood values for a given channel (CPU version)
__host__ __device__ void fetchVals(uchar* input, int in_w, int in_h, int channels, int gxi, int gyi, int c, float vals[4][4]) {
    for (int m = -1; m <= 2; m++) {
        for (int n = -1; n <= 2; n++) {
            int px = clamp(gxi + m, 0, in_w - 1);
            int py = clamp(gyi + n, 0, in_h - 1);
            vals[m + 1][n + 1] = input[(py * in_w + px) * channels + c];
        }
    }
}

// Fetch using texture memory (GPU version)
__device__ void fetchValsTex(hipTextureObject_t* texObjs, int in_w, int in_h, int gxi, int gyi, int c, float vals[4][4]) {
    for (int m = -1; m <= 2; m++) {
        for (int n = -1; n <= 2; n++) {
            int px = clamp(gxi + m, 0, in_w - 1);
            int py = clamp(gyi + n, 0, in_h - 1);
            vals[m + 1][n + 1] = tex2D<unsigned char>(texObjs[c], px, py);
        }
    }
}
    }
}

// Compute bicubic interpolated value at (gx, gy) for channel c
__host__ __device__ float getBicubicValue(uchar* input, int in_w, int in_h, int channels, float gx, float gy, int c) {
    int gxi = (int)gx;
    int gyi = (int)gy;

    float vals[4][4];
    fetchVals(input, in_w, in_h, channels, gxi, gyi, c, vals);

    float tx = gx - gxi;
    float ty = gy - gyi;
    return perform_interpolation(vals, tx, ty);
}

// CUDA kernel for bicubic upscaling using texture memory
__global__ void bicubicUpscaleKernel(hipTextureObject_t* texObjs, uchar* output, int in_w, int in_h, int channels, int scale) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in_w || y >= in_h) return;

    int gxi = x;
    int gyi = y;

    for (int c = 0; c < channels; c++) {
        float vals[4][4];
        fetchValsTex(texObjs, in_w, in_h, gxi, gyi, c, vals);

        for (int i = 0; i < scale; i++) {
            for (int j = 0; j < scale; j++) {
                float gx = (float)x + (float)i / (float)scale;
                float gy = (float)y + (float)j / (float)scale;
                float tx = gx - gxi;
                float ty = gy - gyi;

                float value = perform_interpolation(vals, tx, ty);

                int out_idx = ((y * scale + j) * (in_w * scale) + (x * scale + i)) * channels + c;
                output[out_idx] = (uchar)value;
            }
        }
    }
}

// Macro for CUDA error checking
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            return -1; \
        } \
    } while (0)

// Main function: performs bicubic upscaling using CUDA if available, else CPU
int main(int argc, char** argv) {
    if (argc > 4) {
        std::cerr << "Usage: ./upscale [input_file] [output_file] [scale]\n";
        return -1;
    }
    std::string input_file = (argc > 1) ? argv[1] : "input_tile.jpg";
    std::string output_file = (argc > 2) ? argv[2] : "output_tile.jpg";
    int scale = 2;
    if (argc > 3) {
        try {
            scale = std::stoi(argv[3]);
        } catch (const std::exception& e) {
            std::cerr << "Error: Invalid scale value provided: " << argv[3] << std::endl;
            return -1;
        }
    }
    if (scale <= 1) {
        std::cerr << "Error: scale must be greater than 1\n";
        return -1;
    }

    // Load the input image
    cv::Mat input = cv::imread(input_file);
    if (input.empty()) {
        std::cerr << "Error: Could not load image from " << input_file << "!" << std::endl;
        return -1;
    }

    // Get image properties
    int in_w = input.cols;
    int in_h = input.rows;
    int channels = input.channels();
    int out_w = in_w * scale;
    int out_h = in_h * scale;

    // Validate inputs
    if (channels < 1 || channels > 4) {
        std::cerr << "Error: Unsupported number of channels: " << channels << std::endl;
        return -1;
    }

    // Create output image
    cv::Mat output(out_h, out_w, input.type());

    // Check for GPU availability
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    bool useGPU = deviceCount > 0;

    // Allocate memory: use unified memory for output, textures for input on GPU
    uchar *d_output;
    size_t output_size = out_w * out_h * channels * sizeof(uchar);
    hipTextureObject_t texObjs[4]; // max 4 channels
    hipArray* cuArrays[4];

    if (useGPU) {
        // Split input into channels and create textures
        std::vector<cv::Mat> channelMats;
        cv::split(input, channelMats);

        for (int c = 0; c < channels; c++) {
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
            CUDA_CHECK(hipMallocArray(&cuArrays[c], &channelDesc, in_w, in_h));
            CUDA_CHECK(hipMemcpy2DToArray(cuArrays[c], 0, 0, channelMats[c].data, in_w, in_w, in_h, hipMemcpyHostToDevice));

            hipResourceDesc resDesc = {};
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = cuArrays[c];

            hipTextureDesc texDesc = {};
            texDesc.addressMode[0] = hipAddressModeClamp;
            texDesc.addressMode[1] = hipAddressModeClamp;
            texDesc.filterMode = hipFilterModePoint;
            texDesc.readMode = hipReadModeElementType;

            CUDA_CHECK(hipCreateTextureObject(&texObjs[c], &resDesc, &texDesc, nullptr));
        }

        CUDA_CHECK(hipMallocManaged(&d_output, output_size));
    } else {
        d_output = output.data;
    }

    if (useGPU) {
        // Launch CUDA kernel with 16x16 thread blocks
        dim3 blockDim(16, 16);
        dim3 gridDim((in_w + blockDim.x - 1) / blockDim.x, (in_h + blockDim.y - 1) / blockDim.y);
        bicubicUpscaleKernel<<<gridDim, blockDim>>>(texObjs, d_output, in_w, in_h, channels, scale);

        // Synchronize and check for kernel errors
        CUDA_CHECK(hipDeviceSynchronize());

        // Copy result back to host
        memcpy(output.data, d_output, output_size);
    } else {
        // CPU fallback with OpenMP parallelization
        #pragma omp parallel for
        for (int y_out = 0; y_out < out_h; y_out++) {
            for (int x_out = 0; x_out < out_w; x_out++) {
                float gx = (float)x_out / (float)scale;
                float gy = (float)y_out / (float)scale;
                for (int c = 0; c < channels; c++) {
                    float value = getBicubicValue((uchar*)input.data, in_w, in_h, channels, gx, gy, c);
                    int out_idx = (y_out * out_w + x_out) * channels + c;
                    d_output[out_idx] = (uchar)value;
                }
            }
        }
    }

    // Free memory if GPU
    if (useGPU) {
        for (int c = 0; c < channels; c++) {
            CUDA_CHECK(hipDestroyTextureObject(texObjs[c]));
            CUDA_CHECK(hipFreeArray(cuArrays[c]));
        }
        CUDA_CHECK(hipFree(d_output));
    }

    // Save output image
    cv::imwrite(output_file, output);

    std::cout << "Upscaling complete. Output saved to " << output_file << std::endl;

    return 0;
}
