#include "hip/hip_runtime.h"
/**
 * Bicubic Image Upscaling Tool
 *
 * This program performs 2x bicubic upscaling on images using CUDA for GPU acceleration
 * or falls back to CPU with OpenMP parallelization. Bicubic interpolation provides
 * high-quality upscaling by considering a 4x4 neighborhood of pixels around each
 * target location and applying cubic interpolation in both x and y directions.
 *
 * Usage: ./upscaler [input_file] [output_file] [scale]
 */

#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <exception>
#include <cstring>
#include <cmath>
#ifdef _OPENMP
#include <omp.h>
#endif

// Clamp function for readability
__host__ __device__ inline float clamp(float value, float min_val, float max_val) {
    return fminf(fmaxf(value, min_val), max_val);
}

// Cubic interpolation function for 1D (Catmull-Rom spline)
__host__ __device__ float cubicInterpolate(float p0, float p1, float p2, float p3, float t) {
    return p1 + 0.5f * t * (2.0f * p0 - 5.0f * p1 + 4.0f * p2 - p3 + t * (3.0f * (p1 - p2) + p3 - p0));
}

// Perform bicubic interpolation on a pre-fetched 4x4 neighborhood
__host__ __device__ float perform_interpolation(const float vals[4][4], float tx, float ty) {
    float col[4];
    for (int m = 0; m < 4; m++) {
        col[m] = cubicInterpolate(vals[m][0], vals[m][1], vals[m][2], vals[m][3], tx);
    }
    float value = cubicInterpolate(col[0], col[1], col[2], col[3], ty);
    return clamp(value, 0.0f, 255.0f);
}

// Fetch the 4x4 neighborhood values for a given channel
__host__ __device__ void fetchVals(uchar* input, int in_w, int in_h, int gxi, int gyi, int c, float vals[4][4]) {
    for (int m = -1; m <= 2; m++) {
        for (int n = -1; n <= 2; n++) {
            int px = min(max(gxi + m, 0), in_w - 1);
            int py = min(max(gyi + n, 0), in_h - 1);
            vals[m + 1][n + 1] = input[(py * in_w + px) * 3 + c];
        }
    }
}



// Compute bicubic interpolated value at (gx, gy) for channel c
__host__ __device__ float getBicubicValue(uchar* input, int in_w, int in_h, float gx, float gy, int c) {
    int gxi = (int)gx;
    int gyi = (int)gy;

    float vals[4][4];
    fetchVals(input, in_w, in_h, gxi, gyi, c, vals);

    float tx = gx - gxi;
    float ty = gy - gyi;
    return perform_interpolation(vals, tx, ty);
}

// CUDA kernel for bicubic upscaling
__global__ void bicubicUpscaleKernel(uchar* input, uchar* output, int in_w, int in_h, int scale) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in_w || y >= in_h) return;

    for (int c = 0; c < 3; c++) {
        for (int i = 0; i < scale; i++) {
            for (int j = 0; j < scale; j++) {
                float gx = (float)x + (float)i / (float)scale;
                float gy = (float)y + (float)j / (float)scale;

                float value = getBicubicValue(input, in_w, in_h, gx, gy, c);

                int out_idx = ((y * scale + j) * (in_w * scale) + (x * scale + i)) * 3 + c;
                output[out_idx] = (uchar)value;
            }
        }
    }
}

// Macro for CUDA error checking
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            return -1; \
        } \
    } while (0)

// Main function: performs bicubic upscaling using CUDA if available, else CPU
int main(int argc, char** argv) {
    if (argc > 4) {
        std::cerr << "Usage: ./upscale [input_file] [output_file] [scale]\n";
        return -1;
    }
    std::string input_file = (argc > 1) ? argv[1] : "input_tile.jpg";
    std::string output_file = (argc > 2) ? argv[2] : "output_tile.jpg";
    int scale = 2;
    if (argc > 3) {
        try {
            scale = std::stoi(argv[3]);
        } catch (const std::exception& e) {
            std::cerr << "Error: Invalid scale value provided: " << argv[3] << std::endl;
            return -1;
        }
    }
    if (scale <= 1) {
        std::cerr << "Error: scale must be greater than 1\n";
        return -1;
    }

    // Load the input image
    cv::Mat input = cv::imread(input_file);
    if (input.empty()) {
        std::cerr << "Error: Could not load image from " << input_file << "!" << std::endl;
        return -1;
    }

    // Get image properties
    int in_w = input.cols;
    int in_h = input.rows;
    int out_w = in_w * scale;
    int out_h = in_h * scale;

    // Create output image
    cv::Mat output(out_h, out_w, input.type());

    // Check for GPU availability
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    bool useGPU = deviceCount > 0;
    std::cout << "Using " << (useGPU ? "GPU" : "CPU") << " for upscaling" << std::endl;

    // Allocate memory
    uchar *d_input, *d_output;
    size_t input_size = in_w * in_h * 3 * sizeof(uchar);
    size_t output_size = out_w * out_h * 3 * sizeof(uchar);

    if (useGPU) {
        CUDA_CHECK(hipMalloc(&d_input, input_size));
        CUDA_CHECK(hipMalloc(&d_output, output_size));
        CUDA_CHECK(hipMemcpy(d_input, input.data, input_size, hipMemcpyHostToDevice));

        // Launch CUDA kernel with 16x16 thread blocks
        dim3 blockDim(16, 16);
        dim3 gridDim((in_w + blockDim.x - 1) / blockDim.x, (in_h + blockDim.y - 1) / blockDim.y);
        bicubicUpscaleKernel<<<gridDim, blockDim>>>(d_input, d_output, in_w, in_h, scale);

        // Synchronize and check for kernel errors
        CUDA_CHECK(hipDeviceSynchronize());

        // Copy result back to host
        CUDA_CHECK(hipMemcpy(output.data, d_output, output_size, hipMemcpyDeviceToHost));

        // Free device memory
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_output));
    } else {
        // CPU fallback with OpenMP parallelization
        d_input = input.data;
        d_output = output.data;
        #pragma omp parallel for
        for (int y_out = 0; y_out < out_h; y_out++) {
            for (int x_out = 0; x_out < out_w; x_out++) {
                float gx = (float)x_out / (float)scale;
                float gy = (float)y_out / (float)scale;
                for (int c = 0; c < 3; c++) {
                    float value = getBicubicValue(d_input, in_w, in_h, gx, gy, c);
                    int out_idx = (y_out * out_w + x_out) * 3 + c;
                    d_output[out_idx] = (uchar)value;
                }
            }
        }
    }

    // Save output image
    cv::imwrite(output_file, output);

    std::cout << "Upscaling complete. Output saved to " << output_file << std::endl;

    return 0;
}
