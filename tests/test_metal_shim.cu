#include "hip/hip_runtime.h"
#include "cuda_shim.h"
#include <cassert>
#include <chrono>
#include <iostream>
#include <vector>

// Simple vector addition kernel
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    C[i] = A[i] + B[i];
  }
}

void verifyResults(const float *A, const float *B, const float *C,
                   int numElements) {
  for (int i = 0; i < numElements; i++) {
    float expected = A[i] + B[i];
    if (fabs(C[i] - expected) > 1e-5) {
      printf("Mismatch at element %d: expected %f, got %f\n", i, expected,
             C[i]);
      assert(false);
    }
  }
}

int main() {
  // Initialize data size
  const int numElements = 1 << 20; // 1M elements
  size_t size = numElements * sizeof(float);

  // Allocate host memory
  std::vector<float> h_A(numElements);
  std::vector<float> h_B(numElements);
  std::vector<float> h_C(numElements);

  // Initialize host arrays
  for (int i = 0; i < numElements; i++) {
    h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    h_B[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  // Create streams
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);

  // Create events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Copy data to device asynchronously
  hipMemcpyAsync(d_A, h_A.data(), size, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(d_B, h_B.data(), size, hipMemcpyHostToDevice, stream1);

  // Launch kernel on stream1
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

  hipEventRecord(start, stream1);
  vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C,
                                                            numElements);
  hipEventRecord(stop, stream1);

  // Copy result back to host asynchronously
  hipMemcpyAsync(h_C.data(), d_C, size, hipMemcpyDeviceToHost, stream1);

  // Wait for stream1 to complete
  hipStreamSynchronize(stream1);

  // Verify results
  verifyResults(h_A.data(), h_B.data(), h_C.data(), numElements);

  // Print timing information
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Kernel execution time: %f ms\n", milliseconds);

  // Test stream concurrency
  printf("Testing stream concurrency...\n");

  // Split work between two streams
  int halfSize = numElements / 2;
  size_t halfSizeBytes = halfSize * sizeof(float);

  // First half on stream1
  hipMemcpyAsync(d_A, h_A.data(), halfSizeBytes, hipMemcpyHostToDevice,
                  stream1);
  hipMemcpyAsync(d_B, h_B.data(), halfSizeBytes, hipMemcpyHostToDevice,
                  stream1);
  vectorAdd<<<(halfSize + threadsPerBlock - 1) / threadsPerBlock,
              threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C, halfSize);
  hipMemcpyAsync(h_C.data(), d_C, halfSizeBytes, hipMemcpyDeviceToHost,
                  stream1);

  // Second half on stream2
  hipMemcpyAsync(d_A + halfSize, h_A.data() + halfSize, halfSizeBytes,
                  hipMemcpyHostToDevice, stream2);
  hipMemcpyAsync(d_B + halfSize, h_B.data() + halfSize, halfSizeBytes,
                  hipMemcpyHostToDevice, stream2);
  vectorAdd<<<(halfSize + threadsPerBlock - 1) / threadsPerBlock,
              threadsPerBlock, 0, stream2>>>(d_A + halfSize, d_B + halfSize,
                                             d_C + halfSize, halfSize);
  hipMemcpyAsync(h_C.data() + halfSize, d_C + halfSize, halfSizeBytes,
                  hipMemcpyDeviceToHost, stream2);

  // Wait for both streams to complete
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  // Verify results
  verifyResults(h_A.data(), h_B.data(), h_C.data(), numElements);
  printf("Stream concurrency test passed!\n");

  // Test events
  printf("Testing events...\n");
  hipEvent_t event1, event2;
  hipEventCreate(&event1);
  hipEventCreate(&event2);

  // Record events on stream1 and stream2
  hipEventRecord(event1, stream1);
  hipEventRecord(event2, stream2);

  // Wait for both events
  hipEventSynchronize(event1);
  hipEventSynchronize(event2);

  // Check if events are complete
  hipError_t status = hipEventQuery(event1);
  assert(status == hipSuccess);
  status = hipEventQuery(event2);
  assert(status == hipSuccess);

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipEventDestroy(event1);
  hipEventDestroy(event2);

  printf("All tests passed!\n");
  return 0;
}
